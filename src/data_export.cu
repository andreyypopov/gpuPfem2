#include "hip/hip_runtime.h"
#include "data_export.cuh"

DataExport::DataExport(const Mesh2D &mesh, const ParticleHandler2D *particleHandler)
    : mesh(mesh)
    , particleHandler(particleHandler)
    , particleCount(0)
{

}

void DataExport::addScalarDataVector(const deviceVector<double> &dataVector, const std::string &fieldname)
{
    scalarDataVectors[fieldname] = dataVector.data;
    hostScalarDataVectors[fieldname].resize(mesh.getHostVertices().size());
}

void DataExport::exportToVTK(const std::string &filename) const
{
    std::ofstream outputFile(filename.c_str());
    if(outputFile.is_open()){
        const auto &hostVertices = mesh.getHostVertices();
        const auto &hostCells = mesh.getHostCells();

        //header
        outputFile << "<?xml version=\"1.0\" ?> " << std::endl;
        outputFile << "<VTKFile type=\"UnstructuredGrid\" version=\"0.1\" byte_order=\"LittleEndian\">" << std::endl;
        outputFile << "  <UnstructuredGrid>" << std::endl;
        outputFile << "    <Piece NumberOfPoints=\"" << hostVertices.size() <<  "\" NumberOfCells=\"" << hostCells.size() << "\">" << std::endl;

        //vertices
        outputFile << "      <Points>" << std::endl;
        outputFile << "        <DataArray type=\"Float32\" NumberOfComponents=\"3\" Format=\"ascii\">" << std::endl;
        for(const Point2 &vertex : hostVertices)
            outputFile << "        " << vertex.x << " " << vertex.y << " 0.0" << std::endl;
        outputFile << "        </DataArray>" << std::endl;
        outputFile << "      </Points>" << std::endl;

        //cells (triangles)
        outputFile << "      <Cells>" << std::endl;
        outputFile << "        <DataArray type=\"Int32\" Name=\"connectivity\" Format=\"ascii\">" << std::endl;
        outputFile << "          ";
        for (const uint3 &cell : hostCells)
            outputFile << cell.x << " " << cell.y << " " << cell.z << " ";
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        //offsets
        outputFile << "        <DataArray type=\"Int32\" Name=\"offsets\" Format=\"ascii\">" << std::endl;
        outputFile << "          ";
        for (int i = 0; i < hostCells.size(); ++i)
            outputFile << (i + 1) * 3 << " ";
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        //cell types
        outputFile << "        <DataArray type=\"UInt8\" Name=\"types\" Format=\"ascii\">" << std::endl;
        outputFile << "          ";
        for (int i = 0; i < hostCells.size(); ++i)
            outputFile << 5 << " ";
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        outputFile << "      </Cells>" << std::endl;

        if(!scalarDataVectors.empty()){
            outputFile << "      <PointData Scalars=\"scalars\">" << std::endl;

            for(const auto& it : scalarDataVectors){
                outputFile << "        <DataArray type=\"Float32\" Name=\"" << it.first << "\" Format=\"ascii\">" << std::endl;
                outputFile << "        ";

                const double *hostData = hostScalarDataVectors.at(it.first).data();

                copy_d2h(it.second, hostData, hostVertices.size());

                for(int i = 0; i < hostVertices.size(); ++i)
                    outputFile << hostData[i] << "  ";
                outputFile << std::endl;

                outputFile << "        </DataArray>" << std::endl;
            }

            outputFile << "      </PointData>" << std::endl;
        }

        //footer
        outputFile << "    </Piece>" << std::endl;
        outputFile << "  </UnstructuredGrid>" << std::endl;
        outputFile << "</VTKFile>" << std::endl;

        outputFile.close();
        printf("Mesh solution saved to %s\n", filename.c_str());
    } else
        printf("Error while saving mesh solution to a file\n");
}

void DataExport::exportParticlesToVTK(const std::string &filename)
{
    if(particleCount != particleHandler->getParticleCount()){
        particleCount = particleHandler->getParticleCount();
        hostParticles.resize(particleCount);
    }

    copy_d2h(particleHandler->getParticles(), hostParticles.data(), particleCount);

    std::ofstream outputFile(filename.c_str());
    if(outputFile.is_open()){
        //header
        outputFile << "<?xml version=\"1.0\" ?> " << std::endl;
        outputFile << "<VTKFile type=\"UnstructuredGrid\" version=\"0.1\" byte_order=\"LittleEndian\">" << std::endl;
        outputFile << "  <UnstructuredGrid>" << std::endl;
        outputFile << "    <Piece NumberOfPoints=\"" << particleCount <<  "\" NumberOfCells=\"" << particleCount << "\">" << std::endl;

        //positions
    	outputFile << "      <Points>" << std::endl;
	    outputFile << "        <DataArray type=\"Float32\" NumberOfComponents=\"3\" Format=\"ascii\">" << std::endl;
	    for(const auto &particleIndex : hostParticles)
		    outputFile << "          " << particleIndex.getPosition().x << " " << particleIndex.getPosition().y << " 0.0" << std::endl;

	    outputFile << "        </DataArray>" << std::endl;
    	outputFile << "      </Points>" << std::endl;

        //cells (equal to particles)
        outputFile << "      <Cells>" << std::endl;
        outputFile << "        <DataArray type=\"Int32\" Name=\"connectivity\" Format=\"ascii\">" << std::endl;
        outputFile << "        ";
        for (int i = 0; i < particleCount; ++i)
            outputFile << "  " << i;
        
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        //offsets
        outputFile << "        <DataArray type=\"Int32\" Name=\"offsets\" Format=\"ascii\">" << std::endl;
        outputFile << "        ";
        for (int i = 0; i < particleCount; ++i)
            outputFile << "  " << i + 1; 
        
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        outputFile << "        <DataArray type=\"Int32\" Name=\"types\" Format=\"ascii\">" << std::endl;
        outputFile << "        ";
        for (int i = 0; i < particleCount; ++i)
            outputFile << "  " << 1;
        
        outputFile << std::endl;
        outputFile << "        </DataArray>" << std::endl;
        outputFile << "      </Cells>" << std::endl;

        //data in particles (field values)
	    outputFile << "      <PointData Scalars=\"scalars\">" << std::endl;
	
        //velocity
        outputFile << "        <DataArray type=\"Float32\" Name=\"velocity\" NumberOfComponents=\"3\" Format=\"ascii\">" << std::endl;
        for(const auto &particleIndex : hostParticles)
            outputFile << "          " << particleIndex.getVelocity().x << " " << particleIndex.getVelocity().y << " 0.0" << std::endl;
        outputFile << "        </DataArray>" << std::endl;

        outputFile << "      </PointData>" << std::endl;

        //footer
        outputFile << "    </Piece>" << std::endl;
        outputFile << "  </UnstructuredGrid>" << std::endl;
        outputFile << "</VTKFile>" << std::endl;

        outputFile.close();
        printf("Particles saved to %s\n", filename.c_str());
    } else
        printf("Error while saving particles to a file\n");
}
