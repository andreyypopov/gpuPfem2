#include "hip/hip_runtime.h"
#include "particle_handler_2d.cuh"

#include "../geometry.cuh"

__constant__ Point3 subcellCenters[CONSTANTS::MAX_PARTICLES_PER_CELL];
__constant__ int particlesPerCell;

__global__ void kSeedParticlesIntoCell(int n, const Point2 *vertices, const uint3 *cells, Particle2D *particles, int *count){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const uint3 triangle = cells[idx];

        Point2 triangleVertices[3];
        triangleVertices[0] = vertices[triangle.x];
        triangleVertices[1] = vertices[triangle.y];
        triangleVertices[2] = vertices[triangle.z];

        int startIndex = atomicAdd(count, particlesPerCell);
        for(int i = 0; i < particlesPerCell; ++i){
            Particle2D particle(GEOMETRY::transformLocalToGlobal(subcellCenters[i], triangleVertices), subcellCenters[i], startIndex + i);
            particle.setCellID(idx);
            particles[startIndex + i] = particle;
        }
    }
}

__global__ void kAdvectParticles(int n, const uint3 *cells, Particle2D *particles, double **velocity, double timeStep){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        Particle2D &particle = particles[idx];

        const uint3 triangle = cells[particle.getCellID()];

        const Point3 localPos = particles[idx].getLocalPosition();
        Point2 advectionVelocity = { 0.0, 0.0 };
        for(int i = 0; i < 3; ++i){
            advectionVelocity.x += *(&localPos.x + i) * velocity[0][*(&triangle.x + i)];
            advectionVelocity.y += *(&localPos.x + i) * velocity[1][*(&triangle.x + i)];
        }

        particle.setPosition(particle.getPosition() + timeStep * advectionVelocity);
    }
}

__global__ void kCorrectParticleVelocity(int n, const uint3 *cells, Particle2D *particles, double **velocity, double **velocityOld = nullptr){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        Particle2D &particle = particles[idx];

        const uint3 triangle = cells[particle.getCellID()];

        const Point3 localPos = particle.getLocalPosition();
        Point2 velocityIncrement = { 0.0, 0.0 };
        for(int i = 0; i < 3; ++i){
            velocityIncrement.x += *(&localPos.x + i) * (velocity[0][*(&triangle.x + i)] - (velocityOld ? velocityOld[0][*(&triangle.x + i)] : 0.0));
            velocityIncrement.y += *(&localPos.x + i) * (velocity[1][*(&triangle.x + i)] - (velocityOld ? velocityOld[1][*(&triangle.x + i)] : 0.0));
        }

        particle.setVelocity(particle.getVelocity() + velocityIncrement);
    }
}

__global__ void kProjectParticleVelocityOntoGrid(int n, const uint3 *cells, Particle2D *particles, double **projectionVelocity, double *projectionWeights){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        Particle2D &particle = particles[idx];

        const uint3 triangle = cells[particle.getCellID()];

        const Point3 localPos = particle.getLocalPosition();
        for(int i = 0; i < 3; ++i){
            const double shapeValue = *(&localPos.x + i);
            const unsigned int index = *(&triangle.x + i);
            
            atomicAdd(&projectionVelocity[0][index], shapeValue * particle.getVelocity().x);
            atomicAdd(&projectionVelocity[1][index], shapeValue * particle.getVelocity().y);
            atomicAdd(&projectionWeights[index], shapeValue);
        }
    }
}

__global__ void kFinalizeVelocityProjection(int n, double **velocity, double **projectionVelocity, double *projectionWeights){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        for(int i = 0; i < 2; ++i)
            velocity[i][idx] = projectionVelocity[i][idx] / projectionWeights[idx];
    }
}

__global__ void kCheckParticleInCell(int n, const Point2 *vertices, const uint3 *cells, const Matrix2x2 *invJacobi, Particle2D *particles, int *particlesForCheckInNeighborsCount, int *particlesForCheckInNeighbors){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        Particle2D &particle = particles[idx];
        const uint3 triangle = cells[particle.getCellID()];

        const Point2 v3 = vertices[triangle.z];
        const Matrix2x2 cellInvJacobi = invJacobi[particle.getCellID()];

        if(particle.isInsideCell(cellInvJacobi, v3)){

        } else {
            int index = atomicAdd(particlesForCheckInNeighborsCount, 1);
            particlesForCheckInNeighbors[index] = idx;
        }
    }
}

__global__ void kCheckParticleInNeighbors(int n, const Point2 *vertices, const uint3 *cells, const Matrix2x2 *invJacobi, int *cellNeighborOffsets, int *cellNeighborIndices,
    Particle2D *particles, int *particlesForCheckInNeighbors, int *particlesToBeDeletedCount, int *particlesToBeDeleted)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        int particleIdx = particlesForCheckInNeighbors[idx];
        Particle2D &particle = particles[particleIdx];

        const unsigned int oldCellID = particle.getCellID();
        bool foundCell = false;

        for(int neighborIdx = cellNeighborOffsets[oldCellID]; neighborIdx < cellNeighborOffsets[oldCellID + 1]; ++neighborIdx){
            const int neighborID = cellNeighborIndices[neighborIdx];
            
            const Point2 v3 = vertices[cells[neighborID].z];
            const Matrix2x2 cellInvJacobi = invJacobi[neighborID];

            if(particle.isInsideCell(cellInvJacobi, v3)){
                particle.setCellID(neighborID);
                foundCell = true;
                break;
            }
        }

        if(!foundCell){
            int index = atomicAdd(particlesToBeDeletedCount, 1);
            particlesToBeDeleted[index] = particleIdx;
        }
    }
}

__global__ void kDeleteParticles(int n, Particle2D *particles, int *particleCount, int *particlesToBeDeleted)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        int particleIdx = particlesToBeDeleted[idx];
        particles[particleIdx] = particles[*particleCount - n + idx];
    }
}

ParticleHandler2D::ParticleHandler2D(const Mesh2D *mesh_, int cellDivisionLevel)
    : mesh(mesh_)
{
    const int subcellsNumber = std::max(std::min(cellDivisionLevel, CONSTANTS::MAX_CELL_DIVISION_LEVEL), 1);
    const int hostParticlesPerCell = subcellsNumber * subcellsNumber;
    copy_h2const(&hostParticlesPerCell, &particlesPerCell, 1);

    std::vector<Point3> hostSubcellCenters(hostParticlesPerCell);

    int particleNum = -1;
    Point3 subcellVertices[3];
    Point3 center;
    double xmin, xmax, ymin, ymax;
    const double dx = 1.0 / subcellsNumber;
    
    for(int i = 0; i < subcellsNumber; ++i)
        for(int j = 0; j < 2 * i + 1; ++j){
            xmin = (j / 2) * dx;
            xmax = xmin + dx;
            ymin = (subcellsNumber - 1 - i) * dx;
            ymax = ymin + dx;

            subcellVertices[0] = { xmin, ymax, 1.0 - xmin - ymax };
            subcellVertices[1].x = (j % 2 == 0) ? xmin : xmax;
            subcellVertices[1].y = (j % 2 == 0) ? ymin : ymax;
            subcellVertices[1].z = 1.0 - subcellVertices[1].x - subcellVertices[1].y;
            subcellVertices[2] = { xmax, ymin, 1.0 - xmax - ymin };
            
            center = CONSTANTS::ONE_THIRD * (subcellVertices[0] + subcellVertices[1] + subcellVertices[2]);

            hostSubcellCenters[++particleNum] = center;
        }

    copy_h2const(hostSubcellCenters.data(), subcellCenters, hostParticlesPerCell);
    particleCount = hostParticlesPerCell * mesh->getCells().size;

    allocate_device(&deviceParticleCount, 1);
    allocate_device(&particlesForCheckInNeighborCellsCount, 1);
    allocate_device(&particlesToBeDeletedCount, 1);

    for(int i = 0; i < 2; ++i)
        projectionVelocity[i].allocate(mesh->getVertices().size);
    projectionWeights.allocate(mesh->getVertices().size);

    projectionVelocityPtrs.allocate(2);
    double* hostProjectionVelocityPtrs[2];
    for(int i = 0; i < 2; ++i)
        hostProjectionVelocityPtrs[i] = projectionVelocity[i].data;
    
    copy_h2d(hostProjectionVelocityPtrs, projectionVelocityPtrs.data, 2);
}

ParticleHandler2D::~ParticleHandler2D()
{
    free_device(deviceParticleCount);
    free_device(particlesForCheckInNeighborCellsCount);
    free_device(particlesToBeDeletedCount);
}

void ParticleHandler2D::seedParticles()
{
    particles.allocate(particleCount);
    particlesForCheckInNeighborCells.allocate(particleCount);
    particlesToBeDeleted.allocate(particleCount / 10);

    unsigned int blocks = blocksForSize(mesh->getCells().size);

    zero_value_device(deviceParticleCount, 1);
    kSeedParticlesIntoCell<<<blocks, gpuThreads>>>(mesh->getCells().size, mesh->getVertices().data, mesh->getCells().data, particles.data, deviceParticleCount);

    hipDeviceSynchronize();
    int particlesSeeded;
    copy_d2h(deviceParticleCount, &particlesSeeded, 1);

    printf("Created %d particles\n", particlesSeeded);
}

void ParticleHandler2D::initParticleVelocity(const deviceVector<double*> &velocitySolution)
{
    unsigned int blocks = blocksForSize(particleCount);
    kCorrectParticleVelocity<<<blocks, gpuThreads>>>(particleCount, mesh->getCells().data, particles.data, velocitySolution.data);
}

void ParticleHandler2D::advectParticles(const deviceVector<double *> &velocitySolution, double timeStep, int particleSubsteps)
{
    const double particleAdvectionTimeStep = timeStep / particleSubsteps;

    unsigned int blocks;
    for(int i = 0; i < particleSubsteps; ++i){
        blocks = blocksForSize(particleCount);
        kAdvectParticles<<<blocks, gpuThreads>>>(particleCount, mesh->getCells().data, particles.data, velocitySolution.data, particleAdvectionTimeStep);

        sortParticlesInCells();
    }
}

void ParticleHandler2D::correctParticleVelocity(const deviceVector<double *> &velocitySolution, const deviceVector<double *> &velocitySolutionOld)
{
    unsigned int blocks = blocksForSize(particleCount);
    kCorrectParticleVelocity<<<blocks, gpuThreads>>>(particleCount, mesh->getCells().data, particles.data, velocitySolution.data, velocitySolutionOld.data);
}

void ParticleHandler2D::projectVelocityOntoGrid(deviceVector<double *> &velocity)
{
    for(int i = 0; i < 2; ++i)
        zero_value_device(projectionVelocity[i].data, mesh->getVertices().size);
    zero_value_device(projectionWeights.data, mesh->getVertices().size);
    
    unsigned int blocks = blocksForSize(particleCount);
    kProjectParticleVelocityOntoGrid<<<blocks, gpuThreads>>>(particleCount, mesh->getCells().data, particles.data, projectionVelocityPtrs.data, projectionWeights.data);

    blocks = blocksForSize(mesh->getVertices().size);
    kFinalizeVelocityProjection<<<blocks, gpuThreads>>>(mesh->getVertices().size, velocity.data, projectionVelocityPtrs.data, projectionWeights.data);
}

void ParticleHandler2D::sortParticlesInCells()
{
    zero_value_device(particlesForCheckInNeighborCellsCount, 1);
    zero_value_device(particlesToBeDeletedCount, 1);

    //1. Check whether each particle has stayed in the same cell as previously
    unsigned int blocks = blocksForSize(particleCount);
    kCheckParticleInCell<<<blocks, gpuThreads>>>(particleCount, mesh->getVertices().data, mesh->getCells().data, mesh->getInvJacobi().data,
        particles.data, particlesForCheckInNeighborCellsCount, particlesForCheckInNeighborCells.data);

    int hostParticlesForCheckInNeighborsCount;
    copy_d2h(particlesForCheckInNeighborCellsCount, &hostParticlesForCheckInNeighborsCount, 1);

    //2. For those particles which have left the cell check the neighboring cells
    blocks = blocksForSize(hostParticlesForCheckInNeighborsCount);
    kCheckParticleInNeighbors<<<blocks, gpuThreads>>>(hostParticlesForCheckInNeighborsCount, mesh->getVertices().data, mesh->getCells().data, mesh->getInvJacobi().data,
        mesh->getCellNeighborsOffsets().data, mesh->getCellNeighborIndices().data, particles.data, particlesForCheckInNeighborCells.data,
        particlesToBeDeletedCount, particlesToBeDeleted.data);
    
    int hostParticlesToBeDeletedCount;
    copy_d2h(particlesToBeDeletedCount, &hostParticlesToBeDeletedCount, 1);

    //3. Delete the particles for which a cell was not found (it is done by moving them to the end of the particle vector and reducing its length)
    if(hostParticlesToBeDeletedCount){
        blocks = blocksForSize(hostParticlesToBeDeletedCount);
        kDeleteParticles<<<blocks, gpuThreads>>>(hostParticlesToBeDeletedCount, particles.data, deviceParticleCount, particlesToBeDeleted.data);

        particleCount -= hostParticlesToBeDeletedCount;
        copy_h2d(&particleCount, deviceParticleCount, 1);
    }
}
