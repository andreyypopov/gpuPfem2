#include "hip/hip_runtime.h"
#include "preconditioners.cuh"

__global__ void extractDiagonal(int n, double *invDiagonal, const int *rowPtr, const int *colIndex, const double *matrixVal)
{
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= n)
        return;

    const int startElem = rowPtr[row];
    const int endElem = rowPtr[row + 1];
    int diagIndex = indexBinarySearch(row, colIndex + startElem, endElem - startElem);
    if(diagIndex >= 0)
        invDiagonal[row] = 1.0 / matrixVal[startElem + diagIndex];
}

__global__ void applyJacobiPreconditioner(int n, double *dest, const double *src, const double *preconditioner)
{
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= n)
        return;

    dest[row] = src[row] * preconditioner[row];
}

Preconditioner::Preconditioner(int n_, const LinearAlgebra *LA_)
    : n(n_)
    , LA(LA_)
{
    gpuBlocks = blocksForSize(n);
}

PreconditionerJacobi::PreconditionerJacobi(int n_, const LinearAlgebra *LA_)
    : Preconditioner(n_, LA_)
{
    invDiagValues.allocate(n_);
}

void PreconditionerJacobi::initialize(const SparseMatrixCSR &csrMatrix)
{
    extractDiagonal<<<gpuBlocks, gpuThreads>>>(n, invDiagValues.data, csrMatrix.getRowOffset(), csrMatrix.getColIndices(), csrMatrix.getMatrixValues());
}

void PreconditionerJacobi::applyPreconditioner(double *dest, const double *src)
{
    applyJacobiPreconditioner<<<gpuBlocks, gpuThreads>>>(n, dest, src, invDiagValues.data);
}

PreconditionerILU::PreconditionerILU(const SparseMatrixCSR &matrix, const LinearAlgebra *LA_)
    : Preconditioner(matrix.getRows(), LA_)
    , nnz(matrix.getTotalElements())
    , alpha(1.0)
{
    checkCusparseErrors(hipsparseCreateCsrilu02Info(&iluInfo));

    checkCusparseErrors(hipsparseSpSV_createDescr(&lSpsvDescription));
    checkCusparseErrors(hipsparseSpSV_createDescr(&uSpsvDescription));

    checkCusparseErrors(hipsparseCreateMatDescr(&iluMatrix));
    checkCusparseErrors(hipsparseSetMatIndexBase(iluMatrix, HIPSPARSE_INDEX_BASE_ZERO));
    checkCusparseErrors(hipsparseSetMatType(iluMatrix, HIPSPARSE_MATRIX_TYPE_GENERAL));

    matrixValues.allocate(nnz);

    checkCusparseErrors(hipsparseCreateCsr(&lMatrix, n, n, nnz,
        matrix.getRowOffset(), matrix.getColIndices(), matrixValues.data,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    checkCusparseErrors(hipsparseCreateCsr(&uMatrix, n, n, nnz,
        matrix.getRowOffset(), matrix.getColIndices(), matrixValues.data,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseFillMode_t lFillMode = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseFillMode_t uFillMode = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t lDiagType = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseDiagType_t uDiagType = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    checkCusparseErrors(hipsparseSpMatSetAttribute(lMatrix, HIPSPARSE_SPMAT_FILL_MODE, &lFillMode, sizeof(lFillMode)));
    checkCusparseErrors(hipsparseSpMatSetAttribute(uMatrix, HIPSPARSE_SPMAT_FILL_MODE, &uFillMode, sizeof(uFillMode)));
    checkCusparseErrors(hipsparseSpMatSetAttribute(lMatrix, HIPSPARSE_SPMAT_DIAG_TYPE, &lDiagType, sizeof(lDiagType)));
    checkCusparseErrors(hipsparseSpMatSetAttribute(uMatrix, HIPSPARSE_SPMAT_DIAG_TYPE, &uDiagType, sizeof(uDiagType)));

    auxVector.allocate(n);
    //at this point all vector description are set to the auxiliary vector
    //(will be later updated for src and dest vectors at the stage of preconditioner initialization)
    checkCusparseErrors(hipsparseCreateDnVec(&auxVec, n, auxVector.data, HIP_R_64F));
    checkCusparseErrors(hipsparseCreateDnVec(&destVec, n, auxVector.data, HIP_R_64F));
    checkCusparseErrors(hipsparseCreateDnVec(&srcVec, n, auxVector.data, HIP_R_64F));

    const int iluBufferSize = LA->incompleteLU_bufferSize(iluMatrix, matrix.getRowOffset(), matrix.getColIndices(), matrix.getMatrixValues(), iluInfo, n, nnz);
    const int lBuffersize = LA->solveSparseTriangularSystem_bufferSize(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription);
    const int uBufferSize = LA->solveSparseTriangularSystem_bufferSize(uMatrix, auxVec, destVec, &alpha, uSpsvDescription);

    checkCudaErrors(hipMalloc(&iluBuffer, iluBufferSize));
    checkCudaErrors(hipMalloc(&lSpsvBuffer, lBuffersize));
    checkCudaErrors(hipMalloc(&uSpsvBuffer, uBufferSize));
}

PreconditionerILU::~PreconditionerILU()
{
    checkCudaErrors(hipFree(iluBuffer));
    checkCudaErrors(hipFree(lSpsvBuffer));
    checkCudaErrors(hipFree(uSpsvBuffer));
    checkCusparseErrors(hipsparseDestroyCsrilu02Info(iluInfo));
    checkCusparseErrors(hipsparseSpSV_destroyDescr(lSpsvDescription));
    checkCusparseErrors(hipsparseSpSV_destroyDescr(uSpsvDescription));

    checkCusparseErrors(hipsparseDestroyMatDescr(iluMatrix));
    checkCusparseErrors(hipsparseDestroySpMat(lMatrix));
    checkCusparseErrors(hipsparseDestroySpMat(uMatrix));
    checkCusparseErrors(hipsparseDestroyDnVec(auxVec));
    checkCusparseErrors(hipsparseDestroyDnVec(destVec));
    checkCusparseErrors(hipsparseDestroyDnVec(srcVec));
}

void PreconditionerILU::initialize(const SparseMatrixCSR &csrMatrix)
{
    copy_d2d(csrMatrix.getMatrixValues(), matrixValues.data, nnz);

    LA->incompleteLU(iluMatrix, csrMatrix.getRowOffset(), csrMatrix.getColIndices(), matrixValues.data, iluInfo, n, nnz, iluBuffer);
    analysisRequired = true;
}

void PreconditionerILU::applyPreconditioner(double *dest, const double *src)
{
    if(analysisRequired){
        checkCusparseErrors(hipsparseDnVecSetValues(srcVec, (void*)src));
        checkCusparseErrors(hipsparseDnVecSetValues(destVec, (void*)dest));

        LA->solveSparseTriangularSystem_analysis(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription, lSpsvBuffer);
        LA->solveSparseTriangularSystem_analysis(uMatrix, auxVec, destVec, &alpha, uSpsvDescription, uSpsvBuffer);
    }

    LA->solveSparseTriangularSystem(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription);
    LA->solveSparseTriangularSystem(uMatrix, auxVec, destVec, &alpha, uSpsvDescription);
}

PreconditionerIC::PreconditionerIC(const SparseMatrixCSR &matrix, const LinearAlgebra *LA_)
    : Preconditioner(matrix.getRows(), LA_)
    , nnz(matrix.getTotalElements())
    , alpha(1.0)
{
    checkCusparseErrors(hipsparseCreateCsric02Info(&icInfo));

    checkCusparseErrors(hipsparseSpSV_createDescr(&lSpsvDescription));
    checkCusparseErrors(hipsparseSpSV_createDescr(&ltSpsvDescription));

    checkCusparseErrors(hipsparseCreateMatDescr(&icMatrix));
    checkCusparseErrors(hipsparseSetMatIndexBase(icMatrix, HIPSPARSE_INDEX_BASE_ZERO));
    checkCusparseErrors(hipsparseSetMatType(icMatrix, HIPSPARSE_MATRIX_TYPE_GENERAL));

    matrixValues.allocate(nnz);

    checkCusparseErrors(hipsparseCreateCsr(&lMatrix, n, n, nnz,
        matrix.getRowOffset(), matrix.getColIndices(), matrixValues.data,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseFillMode_t lFillMode = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t lDiagType = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    checkCusparseErrors(hipsparseSpMatSetAttribute(lMatrix, HIPSPARSE_SPMAT_FILL_MODE, &lFillMode, sizeof(lFillMode)));
    checkCusparseErrors(hipsparseSpMatSetAttribute(lMatrix, HIPSPARSE_SPMAT_DIAG_TYPE, &lDiagType, sizeof(lDiagType)));

    auxVector.allocate(n);
    //at this point all vector description are set to the auxiliary vector
    //(will be later updated for src and dest vectors at the stage of preconditioner initialization)
    checkCusparseErrors(hipsparseCreateDnVec(&auxVec, n, auxVector.data, HIP_R_64F));
    checkCusparseErrors(hipsparseCreateDnVec(&destVec, n, auxVector.data, HIP_R_64F));
    checkCusparseErrors(hipsparseCreateDnVec(&srcVec, n, auxVector.data, HIP_R_64F));

    const int icBufferSize = LA->incompleteCholesky_bufferSize(icMatrix, matrix.getRowOffset(), matrix.getColIndices(), matrix.getMatrixValues(), icInfo, n, nnz);
    const int lBuffersize = LA->solveSparseTriangularSystem_bufferSize(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription);
    const int ltBufferSize = LA->solveSparseTriangularSystem_bufferSize(lMatrix, auxVec, destVec, &alpha, ltSpsvDescription, true);

    checkCudaErrors(hipMalloc(&icBuffer, icBufferSize));
    checkCudaErrors(hipMalloc(&lSpsvBuffer, lBuffersize));
    checkCudaErrors(hipMalloc(&ltSpsvBuffer, ltBufferSize));
}

PreconditionerIC::~PreconditionerIC()
{
    checkCudaErrors(hipFree(icBuffer));
    checkCudaErrors(hipFree(lSpsvBuffer));
    checkCudaErrors(hipFree(ltSpsvBuffer));
    checkCusparseErrors(hipsparseDestroyCsric02Info(icInfo));
    checkCusparseErrors(hipsparseSpSV_destroyDescr(lSpsvDescription));
    checkCusparseErrors(hipsparseSpSV_destroyDescr(ltSpsvDescription));

    checkCusparseErrors(hipsparseDestroyMatDescr(icMatrix));
    checkCusparseErrors(hipsparseDestroySpMat(lMatrix));
    checkCusparseErrors(hipsparseDestroyDnVec(auxVec));
    checkCusparseErrors(hipsparseDestroyDnVec(destVec));
    checkCusparseErrors(hipsparseDestroyDnVec(srcVec));
}

void PreconditionerIC::initialize(const SparseMatrixCSR &csrMatrix)
{
    copy_d2d(csrMatrix.getMatrixValues(), matrixValues.data, nnz);

    LA->incompleteCholesky(icMatrix, csrMatrix.getRowOffset(), csrMatrix.getColIndices(), matrixValues.data, icInfo, n, nnz, icBuffer);
    analysisRequired = true;
}

void PreconditionerIC::applyPreconditioner(double *dest, const double *src)
{
    if(analysisRequired){
        checkCusparseErrors(hipsparseDnVecSetValues(srcVec, (void*)src));
        checkCusparseErrors(hipsparseDnVecSetValues(destVec, (void*)dest));

        LA->solveSparseTriangularSystem_analysis(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription, lSpsvBuffer);
        LA->solveSparseTriangularSystem_analysis(lMatrix, auxVec, destVec, &alpha, ltSpsvDescription, ltSpsvBuffer, true);
    }

    LA->solveSparseTriangularSystem(lMatrix, srcVec, auxVec, &alpha, lSpsvDescription);
    LA->solveSparseTriangularSystem(lMatrix, auxVec, destVec, &alpha, ltSpsvDescription, true);
}
